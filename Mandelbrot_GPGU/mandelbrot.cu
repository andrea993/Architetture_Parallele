#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "bmp.h"
#include <sys/time.h>

typedef struct _mandelbrotPars
{
	float mx;
	float Mx;
	float my;
	float My;
	int width;
	int height;
	BITMAP M;
	unsigned maxiter;
}mandelbrotPars;

__global__ void global_mandelbrot(mandelbrotPars pars)
{
	int row = pars.height - blockIdx.y -1;
	int col = blockIdx.x;

	float c_re = blockIdx.x/(pars.width-1.0)*(pars.Mx-pars.mx)+pars.mx;
	float c_im = blockIdx.y/(pars.height-1.0)*(pars.My-pars.my)+pars.my;

	int itr = 0;
	float z_re = 0, z_im = 0, z_re_tmp;

	while (z_re*z_re + z_im*z_im < 2*2 && itr<pars.maxiter)
	{
		z_re_tmp = z_re*z_re - z_im*z_im + c_re;
		z_im = 2*z_re*z_im + c_im;
		z_re = z_re_tmp;

		itr++;
	}
	
	COLORTRIPLE color = {0,0,0};
	if (itr < pars.maxiter)
		color.green = color.blue = itr*255/pars.maxiter;
		
	PIXEL(pars.M, row, col) = color;

}

int main(int argc, char **argv)
{

	if (argc != 2)
	{
		fprintf(stderr,"Usage:\nmandelbrot [IMAGE_WIDTH]\n");
		return -1;
	}

	mandelbrotPars pars;
	pars.mx = -2.5;
	pars.Mx = 1;
	pars.my = -1;
	pars.My = 1;
	pars.maxiter = 1024;
	
	pars.width=atoi(argv[1]);
	pars.height=pars.width*(pars.My-pars.my)/(pars.Mx-pars.mx);

	fprintf(stderr,"%d %d\n",pars.width,pars.height);

	pars.M = CreateEmptyBitmap(pars.height, pars.width);
	hipDeviceSynchronize();
	
	dim3 blockPerGrid(pars.width, pars.height, 1);
	dim3 threadPerBlock(1, 1, 1);

	global_mandelbrot <<< blockPerGrid, threadPerBlock >>> (pars);
	hipDeviceSynchronize();

	FILE* fp = fopen("out.bmp","wb");
	WriteBitmap(pars.M, fp);
	fclose(fp);
	ReleaseBitmapData(&pars.M);;

	return EXIT_SUCCESS;
}
